#include <hip/hip_runtime.h>
#include <iostream>

// Kernel function to multiply two arrays
__global__ void multiply_arrays(const float *A, const float *B, float *C, int n) {
    // Implement the addition here
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n) {
        C[idx] = A[idx] * B[idx];
        printf("Thread %d: A[%d] = %f, B[%d] = %f\n, C[%d] = %f\n", idx, idx, A[idx], idx, B[idx], idx, C[idx]);
    }
}

// Kernal function to sum up array.
__global__ void sum_array(float *array, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    for (int stride = 1; stride < n; stride *= 2) {
        if (idx % (2 * stride) == 0 && (idx + stride) < n) {
            array[idx] += array[idx + stride];
        }
    }
    printf("Thread %d: C[%d] = %f, Total = %f \n", idx, idx, array[idx], array[0]);
    __syncthreads();

}

// Utility function to initialize arrays
void initialize_arrays(float *A, float *B, int n) {
    for (int i = 0; i < n; i++) {
        A[i] = static_cast<float>(i);
        B[i] = static_cast<float>(i);
    }
}

// Host function to run the kernel
void add_arrays_cuda(const float *A, const float *B, float *C, int n) {
    // Allocate device memory

    float *d_A, *d_B, *d_C;
    size_t size = n * sizeof(float); // Size of each array in bytes

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);


    // Copy input arrays to the device

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);


    // Launch kernel with appropriate block and grid size

    int threads_per_block = 256;
    int blocks = (n + threads_per_block + 1)/ threads_per_block;

    multiply_arrays<<<blocks, threads_per_block>>>(d_A, d_B, d_C, n);

    sum_array<<<blocks, threads_per_block>>>(d_C, n);


    // Copy the result array back to the host

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);


    // Free device memory

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

int main() {
    const int N = 1024; // Array size

    // Allocate host memory
    float *A = new float[N];
    float *B = new float[N];
    float *C = new float[N];

    // Initialize input arrays
    initialize_arrays(A, B, N);

    // Perform addition on the GPU
    add_arrays_cuda(A, B, C, N);

    // Display results (optional)
    std::cout << "Results:\n";
    std::cout << C[0] << " ";
    std::cout << std::endl;

    // Free host memory
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
